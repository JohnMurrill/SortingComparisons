/*
* Parallel bitonic sort using CUDA.
* Compile with
* nvcc -arch=sm_11 bitonic_sort.cu
* Based on http://www.tools-of-computing.com/tc/CS/Sorts/bitonic_sort.htm
*/
 

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
 
/* Every thread gets exactly one value in the unsorted array. */
#define THREADS 512 // 2^9
#define BLOCKS 32768 // 2^15
#define NUM_VALS THREADS*BLOCKS
 
void print_elapsed(clock_t start, clock_t stop){
    double elapsed = ((double) (stop - start));// CLOCKS_PER_SEC;
    printf("Elapsed time: %.3fs\n", elapsed);
}
 
float random_float(){
    return (float)rand()/(float)RAND_MAX;
}
 
void array_print(float *arr, int length){
    int i;
    for (i = 0; i < length; ++i) {
    printf("%1.3f ", arr[i]);
}
printf("\n");
}
 
void array_fill(float *arr, int length){
    srand(time(NULL));
    int i;
    for (i = 0; i < length; ++i) {
        arr[i] = random_float();
    }
}
 
__global__ void bitonic_sort_step(float *dev_values, int j, int k){
    unsigned int i, ixj; /* Sorting partners: i and ixj */
    i = threadIdx.x + blockDim.x * blockIdx.x;
    ixj = i^j;
 
    /* The threads with the lowest ids sort the array. */
    if ((ixj)>i) {
        if ((i&k)==0) {
            /* Sort ascending */
            if (dev_values[i]>dev_values[ixj]) {
                /* exchange(i,ixj); */
                float temp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = temp;
            }
        }
    if ((i&k)!=0) {
        /* Sort descending */
        if (dev_values[i]<dev_values[ixj]) {
            /* exchange(i,ixj); */
            float temp = dev_values[i];
            dev_values[i] = dev_values[ixj];
            dev_values[ixj] = temp;
        }
    }
}
}
 
/**
* Inplace bitonic sort using CUDA.
*/
void bitonic_sort(float *values, int numVals){
    float *dev_values;
    size_t size = numVals * sizeof(float);
 
    hipMalloc((void**) &dev_values, size);
    hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);
 
    dim3 blocks(BLOCKS,1); /* Number of blocks */
    dim3 threads(THREADS,1); /* Number of threads */
 
    int j, k;
/* Major step */
    for (k = 2; k <= NUM_VALS; k <<= 1) {
/* Minor step */
        for (j=k>>1; j>0; j=j>>1) {
            bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k);
        }
    }
    hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
    hipFree(dev_values);
}
 
int main(void){
    clock_t start, stop;
 
    //Initialize a 1000 element array to be sorted
    float *values = (float*) malloc( 1000 * sizeof(float));
    array_fill(values, 1000);
 
    start = clock();
    bitonic_sort(values, 1000); /* Inplace */
    stop = clock();
 
    print_elapsed(start, stop);
    free(values);
	
    //Initialize a 10,000 element array to be sorted
    float *Secondvalues = (float*) malloc( 10000 * sizeof(float));
    array_fill(Secondvalues, 10000);
 
    start = clock();
    bitonic_sort(Secondvalues, 10000); /* Inplace */
    stop = clock();
 
    print_elapsed(start, stop);
    free(Secondvalues);

    //Initialize a 1,000,000 element array to be sorted
    float *Thirdvalues = (float*) malloc( 1000000 * sizeof(float));
    array_fill(Thirdvalues, 1000000);
 
    start = clock();
    bitonic_sort(Thirdvalues, 1000000); /* Inplace */
    stop = clock();
 
    print_elapsed(start, stop);
    free(Thirdvalues);

    //Initialize a 2^24 element array to be sorted
    float *fourthvalues = (float*) malloc( NUM_VALS * sizeof(float));
    array_fill(fourthvalues, NUM_VALS);
 
    start = clock();
    bitonic_sort(fourthvalues, NUM_VALS); /* Inplace */
    stop = clock();
 
    print_elapsed(start, stop);
    free(fourthvalues);
}
